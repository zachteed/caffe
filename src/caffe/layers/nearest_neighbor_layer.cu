#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------

#include <cfloat>

#include "caffe/layers/nearest_neighbor_layer.hpp"

using std::max;
using std::min;

namespace caffe {

template <typename Dtype>
__global__ void NearestNeighborForward(const int nthreads, const Dtype* bottom_data,
    const int channels, const int height, const int width,
    const int top_height, const int top_width, Dtype* top_data) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int w = index % top_width;
    int h = (index / top_width) % top_height;
    int c = (index / top_width / top_height) % channels;
    int n = index / top_width / top_height / channels;

    int bottom_index = (n*channels + c) * height * width;
    bottom_index += (h/2) * width + (w/2);
    top_data[index] = bottom_data[bottom_index];
  }
}

template <typename Dtype>
void NearestNeighborLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  NearestNeighborForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, channels_, height_, width_, top_height_, top_width_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void NearestNeighborBackward(const int nthreads, const Dtype* top_diff,
    const int channels, const int height, const int width,
    const int top_height, const int top_width, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, h, w) coords in bottom data
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    Dtype gradient = 0;

    int top_index = (n*channels+c)*top_height*top_width;
    const Dtype *offset_top_diff = top_diff + top_index;


    gradient += offset_top_diff[2*h*top_width + 2*w];
    gradient += offset_top_diff[(2*h+1)*top_width + 2*w];
    gradient += offset_top_diff[2*h*top_width + (2*w+1)];
    gradient += offset_top_diff[(2*h+1)*top_width + (2*w+1)];
    bottom_diff[index] = gradient;

  }
}

template <typename Dtype>
void NearestNeighborLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);

  // NOLINT_NEXT_LINE(whitespace/operators)
  NearestNeighborBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, channels_, height_, width_,
      top_height_, top_width_, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(NearestNeighborLayer);

}  // namespace caffe
